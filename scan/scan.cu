#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"

#define THREADS_PER_BLOCK 256


// helper function to round an integer up to the next power of 2
static inline int nextPow2(int n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

__global__ void 
upsweep(int* result, int two_d) {
    int index = ( blockIdx.x * blockDim.x + threadIdx.x + 1 ) * two_d * 2;
    result[index-1] += result[index-two_d-1];
}

__global__ void 
downsweep(int* result, int two_d) {
    int index = ( blockIdx.x * blockDim.x + threadIdx.x + 1 ) * two_d * 2;
    int t = result[index-two_d-1];
    result[index-two_d-1] = result[index-1];
    result[index-1] += t;
}

// exclusive_scan --
//
// Implementation of an exclusive scan on global memory array `input`,
// with results placed in global memory `result`.
//
// N is the logical size of the input and output arrays, however
// students can assume that both the start and result arrays we
// allocated with next power-of-two sizes as described by the comments
// in cudaScan().  This is helpful, since your parallel scan
// will likely write to memory locations beyond N, but of course not
// greater than N rounded up to the next power of 2.
//
// Also, as per the comments in cudaScan(), you can implement an
// "in-place" scan, since the timing harness makes a copy of input and
// places it in result
void exclusive_scan(int* input, int N, int* result)
{

    // CS149 TODO:
    //
    // Implement your exclusive scan implementation here.  Keep input
    // mind that although the arguments to this function are device
    // allocated arrays, this is a function that is running in a thread
    // on the CPU.  Your implementation will need to make multiple calls
    // to CUDA kernel functions (that you must write) to implement the
    // scan.

    int blocks = 0;
    int total_threads = 0;
    int rounded_length = nextPow2(N);

    // upsweep
    for (int two_d=1; two_d<rounded_length/2; two_d*=2) {
        total_threads = rounded_length / two_d / 2;
        if (total_threads > THREADS_PER_BLOCK) {
            blocks = total_threads / THREADS_PER_BLOCK;
            upsweep<<<blocks, THREADS_PER_BLOCK>>>(result, two_d);
        } else {
            upsweep<<<1, total_threads>>>(result, two_d);
        }
    }
    // result[rounded_length-1] = 0;
    int zero = 0;
    hipMemcpy(result+rounded_length-1, &zero, sizeof(int), hipMemcpyHostToDevice);
    // downsweep
    for (int two_d=rounded_length/2; two_d>=1; two_d/=2) {
        total_threads = rounded_length / two_d / 2;
        if (total_threads <= THREADS_PER_BLOCK) {
            downsweep<<<1, total_threads>>>(result, two_d);
        } else {
            blocks = total_threads / THREADS_PER_BLOCK;
            downsweep<<<blocks, THREADS_PER_BLOCK>>>(result, two_d);
        }
    }

}


//
// cudaScan --
//
// This function is a timing wrapper around the student's
// implementation of scan - it copies the input to the GPU
// and times the invocation of the exclusive_scan() function
// above. Students should not modify it.
double cudaScan(int* inarray, int* end, int* resultarray)
{
    int* device_result;
    int* device_input;
    int N = end - inarray;  

    // This code rounds the arrays provided to exclusive_scan up
    // to a power of 2, but elements after the end of the original
    // input are left uninitialized and not checked for correctness.
    //
    // Student implementations of exclusive_scan may assume an array's
    // allocated length is a power of 2 for simplicity. This will
    // result in extra work on non-power-of-2 inputs, but it's worth
    // the simplicity of a power of two only solution.

    int rounded_length = nextPow2(end - inarray);
    
    hipMalloc((void **)&device_result, sizeof(int) * rounded_length);
    hipMalloc((void **)&device_input, sizeof(int) * rounded_length);

    // For convenience, both the input and output vectors on the
    // device are initialized to the input values. This means that
    // students are free to implement an in-place scan on the result
    // vector if desired.  If you do this, you will need to keep this
    // in mind when calling exclusive_scan from find_repeats.
    hipMemcpy(device_input, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_result, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    exclusive_scan(device_input, N, device_result);

    // Wait for completion
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
       
    hipMemcpy(resultarray, device_result, (end - inarray) * sizeof(int), hipMemcpyDeviceToHost);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}


// cudaScanThrust --
//
// Wrapper around the Thrust library's exclusive scan function
// As above in cudaScan(), this function copies the input to the GPU
// and times only the execution of the scan itself.
//
// Students are not expected to produce implementations that achieve
// performance that is competition to the Thrust version, but it is fun to try.
double cudaScanThrust(int* inarray, int* end, int* resultarray) {

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);
    
    hipMemcpy(d_input.get(), inarray, length * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
   
    hipMemcpy(resultarray, d_output.get(), length * sizeof(int), hipMemcpyDeviceToHost);

    thrust::device_free(d_input);
    thrust::device_free(d_output);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}

__global__ void
repeats_cmp(int* input, int N, int* output) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
        output[i] = input[i] == input[i+1] ? 1 : 0;
}

__global__ void 
repeats_cpy(int* input, int* output) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    output[i] = input[i];
}

__global__ void
repeats_out(int* sum, int* cmp, int* output) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;;
    if (cmp[i]==1) {
        output[sum[i]] = i;
    }
}

// find_repeats --
//
// Given an array of integers `device_input`, returns an array of all
// indices `i` for which `device_input[i] == device_input[i+1]`.
//
// Returns the total number of pairs found
int find_repeats(int* device_input, int length, int* device_output) {

    // CS149 TODO:
    //
    // Implement this function. You will probably want to
    // make use of one or more calls to exclusive_scan(), as well as
    // additional CUDA kernel launches.
    //    
    // Note: As in the scan code, the calling code ensures that
    // allocated arrays are a power of 2 in size, so you can use your
    // exclusive_scan function with them. However, your implementation
    // must ensure that the results of find_repeats are correct given
    // the actual array length.

    int N = length - 1;

    if (N <= THREADS_PER_BLOCK) {
        repeats_cmp<<<1, N>>>(device_input, N, device_input);
        repeats_cpy<<<1, N>>>(device_input, device_output);
        exclusive_scan(device_input, length, device_input);
        repeats_out<<<1, N>>>(device_input, device_output, device_output);
    } else {
        int blocks = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        repeats_cmp<<<blocks, THREADS_PER_BLOCK>>>(device_input, N, device_input);
        repeats_cpy<<<blocks, THREADS_PER_BLOCK>>>(device_input, device_output);
        exclusive_scan(device_input, length, device_input);
        repeats_out<<<blocks, THREADS_PER_BLOCK>>>(device_input, device_output, device_output);
    }
    int* output = new int;
    hipMemcpy(output, device_input+N, sizeof(int), hipMemcpyDeviceToHost);
    int num = *output; 
    delete output;
    return num;
}


//
// cudaFindRepeats --
//
// Timing wrapper around find_repeats. You should not modify this function.
double cudaFindRepeats(int *input, int length, int *output, int *output_length) {

    int *device_input;
    int *device_output;
    int rounded_length = nextPow2(length);
    
    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    double startTime = CycleTimer::currentSeconds();
    
    int result = find_repeats(device_input, length, device_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    // set output count and results array
    *output_length = result;
    hipMemcpy(output, device_output, length * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);

    float duration = endTime - startTime; 
    return duration;
}



void printCudaInfo()
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}
